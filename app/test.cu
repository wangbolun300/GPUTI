#include "hip/hip_runtime.h"
#include <gputi/root_finder.h>
#include <gputi/book.h>
#include "timer.hpp"
#include <iostream>
#include <functional>
#include <fstream>
#include <filesystem>
#include <cuda/std/functional>

#include <gputi/timer.cuh>
#include <gputi/io.h>

using namespace ccd;
extern std::vector<std::string> simulation_folders;
extern std::vector<std::string> handcrafted_folders;

std::array<std::array<Scalar, 3>, 8> substract_ccd(const std::vector<std::array<Scalar, 3>> &data, int nbr)
{
    std::array<std::array<Scalar, 3>, 8> result;
    int start = nbr * 8;
    for (int i = 0; i < 8; i++)
    {
        result[i] = data[i + start];
    }
    return result;
}
void write_summary(
    const std::string file,
    const int method,
    const int total_number,
    const int positives,
    const bool is_edge_edge,
    const int fp,
    const int fn,
    const double average_time,
    const double time_lower,
    const double time_upper)
{
    std::ofstream fout;
    fout.open(file);
    fout << "method,total_nbr,positives,is_edge_edge,fp,fn,average_time, time_lower, time_upper"
         << std::endl;
    fout << method << "," << total_number << "," << positives << ","
         << is_edge_edge << "," << fp << "," << fn << ',' << average_time << "," << time_lower << "," << time_upper
         << std::endl;
    fout.close();
}
template <typename T>
void write_csv(const std::string &file, const std::vector<std::string> titles, const std::vector<T> data, bool large_info)
{
    std::cout << "inside write" << std::endl;
    std::ofstream fout;
    fout.open(file);

    if (large_info)
    {
        fout << "data" << std::endl;
        for (int i = 0; i < data.size(); i++)
        {
            fout << data[i] << std::endl;
        }
    }
    else
    {
        for (int i = 0; i < titles.size() - 1; i++)
        {
            fout << titles[i] << ",";
        }
        fout << titles.back() << std::endl;
        for (int i = 0; i < data.size() - 1; i++)
        {
            fout << data[i] << ",";
        }
        fout << data.back() << std::endl;
    }

    fout.close();
}


__global__ void run_parallel_vf_ccd_all(CCDdata *data,CCDConfig *config_in, bool *res, int size, Scalar *tois
)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx >= size) return;
    // copy the input queries to __device__
    CCDdata data_in;
    for (int i = 0; i < 3; i++)
    {
        data_in.v0s[i] = data[tx].v0s[i];
        data_in.v1s[i] = data[tx].v1s[i];
        data_in.v2s[i] = data[tx].v2s[i];
        data_in.v3s[i] = data[tx].v3s[i];
        data_in.v0e[i] = data[tx].v0e[i];
        data_in.v1e[i] = data[tx].v1e[i];
        data_in.v2e[i] = data[tx].v2e[i];
        data_in.v3e[i] = data[tx].v3e[i];
    }
    // copy the configurations to the shared memory
    __shared__ CCDConfig config;
    config.err_in[0]=config_in->err_in[0];
    config.err_in[1]=config_in->err_in[1];
    config.err_in[2]=config_in->err_in[2];
    config.co_domain_tolerance=config_in->co_domain_tolerance; // tolerance of the co-domain
    config.max_t=config_in->max_t; // the upper bound of the time interval
    config.max_itr=config_in->max_itr;// the maximal nbr of iterations
    CCDOut out;
    vertexFaceCCD(data_in,config, out);
    res[tx] = out.result;
    tois[tx] = out.toi;
}
__global__ void run_parallel_ee_ccd_all(CCDdata *data,CCDConfig *config_in, bool *res, int size, Scalar *tois
)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx >= size) return;
    // copy the input queries to __device__
    CCDdata data_in;
    for (int i = 0; i < 3; i++)
    {
        data_in.v0s[i] = data[tx].v0s[i];
        data_in.v1s[i] = data[tx].v1s[i];
        data_in.v2s[i] = data[tx].v2s[i];
        data_in.v3s[i] = data[tx].v3s[i];
        data_in.v0e[i] = data[tx].v0e[i];
        data_in.v1e[i] = data[tx].v1e[i];
        data_in.v2e[i] = data[tx].v2e[i];
        data_in.v3e[i] = data[tx].v3e[i];
    }
    // copy the configurations to the shared memory
    __shared__ CCDConfig config;
    config.err_in[0]=config_in->err_in[0];
    config.err_in[1]=config_in->err_in[1];
    config.err_in[2]=config_in->err_in[2];
    config.co_domain_tolerance=config_in->co_domain_tolerance; // tolerance of the co-domain
    config.max_t=config_in->max_t; // the upper bound of the time interval
    config.max_itr=config_in->max_itr;// the maximal nbr of iterations
    CCDOut out;
    edgeEdgeCCD(data_in,config, out);
    res[tx] = out.result;
    tois[tx] = out.toi;
}


void all_ccd_run(const std::vector<std::array<std::array<Scalar, 3>, 8>> &V, bool is_edge,
                 std::vector<bool> &result_list, double &run_time, std::vector<Scalar> &time_impact, int parallel_nbr)
{
    int nbr = V.size();
    result_list.resize(nbr);
    // host
    CCDdata *data_list = new CCDdata[nbr];
    for (int i = 0; i < nbr; i++)
    {
        data_list[i] = array_to_ccd( V[i]);
    }
    bool *res = new bool[nbr];
    Scalar *tois = new Scalar[nbr];
    CCDConfig *config=new CCDConfig[1];
    config[0].err_in[0]=-1;// the input error bound calculate from the AABB of the whole mesh
    config[0].co_domain_tolerance=1e-6; // tolerance of the co-domain
    config[0].max_t=1; // the upper bound of the time interval
    config[0].max_itr=1e6;// the maximal nbr of iterations

    // device
    CCDdata *d_data_list;
    bool *d_res;
    Scalar *d_tois;
    CCDConfig *d_config;

    int data_size = sizeof(CCDdata) * nbr;
    int result_size = sizeof(bool) * nbr;
    int time_size = sizeof(Scalar) * nbr;
   // int dbg_size=sizeof(Scalar)*8;

    hipMalloc(&d_data_list, data_size);
    hipMalloc(&d_res, result_size);
    hipMalloc(&d_tois, time_size);
    hipMalloc(&d_config, sizeof(CCDConfig));

    hipMemcpy(d_data_list, data_list, data_size, hipMemcpyHostToDevice);
    hipMemcpy(d_config, config, sizeof(CCDConfig), hipMemcpyHostToDevice);

    ccd::Timer timer;
    hipProfilerStart();
     timer.start();
    if(is_edge){
        run_parallel_ee_ccd_all<<<nbr / parallel_nbr + 1, parallel_nbr>>>( 
        d_data_list,d_config, d_res, nbr, d_tois);
    }
    else{
        run_parallel_vf_ccd_all<<<nbr / parallel_nbr + 1, parallel_nbr>>>( 
        d_data_list,d_config, d_res, nbr, d_tois);
    }
   
    
    hipDeviceSynchronize();
    double tt = timer.getElapsedTimeInMicroSec();
    run_time = tt;
    hipProfilerStop();

    hipMemcpy(res, d_res, result_size, hipMemcpyDeviceToHost);
    hipMemcpy(tois, d_tois, time_size, hipMemcpyDeviceToHost);
    //hipMemcpy(dbg, d_dbg, dbg_size, hipMemcpyDeviceToHost);

    hipFree(d_data_list);
    hipFree(d_res);
    hipFree(d_tois);
    hipFree(d_config);
    //hipFree(d_dbg);

    for (int i = 0; i < nbr; i++)
    {
        result_list[i] = res[i];
    }

    time_impact.resize(nbr);

    for (int i = 0; i < nbr; i++)
    {
        time_impact[i] = tois[i];
    }
    // std::cout << "dbg info\n"
    //           << dbg[0] << "," << dbg[1] << "," << dbg[2] << "," << dbg[3] << "," << dbg[4] << "," << dbg[5] << "," << dbg[6] << "," << dbg[7] << std::endl;
    delete[] res;
    delete[] data_list;
    delete[] tois;
    delete[] config;
    //delete[] dbg;
    hipError_t ct = hipGetLastError();
    printf("******************\n%s\n************\n", hipGetErrorString(ct));
    
    return;
}


bool WRITE_STATISTIC = true;

void run_rational_data_single_method_parallel(
    const Args &args,
    const bool is_edge_edge,
    const bool is_simulation_data, int parallel, const std::string folder = "", const std::string tail = "")
{
    std::vector<std::array<Scalar, 3>> all_V;
    std::vector<bool> results;

    //std::vector<write_format> queryinfo;
    int total_number = -1;
    int total_positives = 0;
    int num_false_positives = 0;
    int num_false_negatives = 0;
    double time_lower = 1e100;
    double time_upper = -1;
    std::string sub_folder = is_edge_edge ? "/edge-edge/" : "/vertex-face/";
    std::string sub_name = is_edge_edge ? "edge-edge" : "vertex-face";
    std::vector<long> queue_sizes;
    std::vector<Scalar> tois;

    std::vector<bool> result_list;
    std::vector<bool> expect_list;
    std::vector<std::array<std::array<Scalar, 3>, 8>> queries;
    const std::vector<std::string> &scene_names = is_simulation_data ? ccd::simulation_folders : ccd::handcrafted_folders;
    std::cout << "loading data" << std::endl;
    std::vector<std::string> bases = file_path_base();
    for (const auto &scene_name : scene_names)
    {
        std::string scene_path = args.data_dir + scene_name + sub_folder;

        bool skip_folder = false;
        for (const auto &entry : bases)
        {
            if (skip_folder)
            {
                break;
            }
            std::string filename = scene_path + sub_name + "-" + entry + ".csv";

            // std::cout<<"filename "<<filename<<std::endl;
            // exit(0);
            if (queries.size() > TEST_NBR_QUERIES)
            {
                break;
            }
            // all_V = ccd::read_rational_csv(filename, results);
            // all_V = read_rational_csv_bin(filename, results);

            std::string filename_noext =  filename.substr(0, filename.find_last_of("."));
            
            std::string vertexFilename = std::string(filename_noext + "_vertex.bin");
            std::ifstream vinfile (vertexFilename, std::ios::in | std::ios::binary);

            std::string resultsFilename = std::string(filename_noext + "_result.bin");
            std::ifstream rinfile (resultsFilename, std::ios::in | std::ios::binary);
            
            if (vinfile && rinfile)
            {
                read_rational_binary(vertexFilename, all_V );
                read_rational_binary(resultsFilename, results );
            }
            else 
                all_V = read_rational_csv_bin(filename, results);
            
            if (all_V.size() == 0)
            {
                std::cout << "data size " << all_V.size() << std::endl;
                std::cout << filename << std::endl;
            }

            if (all_V.size() == 0)
            {
                skip_folder = true;
                continue;
            }

            int v_size = all_V.size() / 8;
            for (int i = 0; i < v_size; i++)
            {
                if (queries.size() > TEST_NBR_QUERIES)
                {
                    break;
                }
                total_number += 1;

                std::array<std::array<Scalar, 3>, 8> V = substract_ccd(all_V, i);
                bool expected_result = results[i * 8];
                queries.push_back(V);
                expect_list.push_back(expected_result);
            }
        }
    }

#ifdef GPUTI_GO_DEAP_HEAP
    std::array<std::array<Scalar, 3>, 8> deep_one = queries[TESTING_ID];
    std::cout<<"query\n";
    for(int i=0;i<8;i++){
        std::cout<<deep_one[i][0]<<", "<<deep_one[i][1]<<", "<<deep_one[i][2]<<std::endl;
        if(i==3){
            std::cout<<std::endl;
        }
    }
    queries.resize(TEST_SIZE);
    expect_list.resize(queries.size());
    for (int i = 0; i < queries.size(); i++)
    {
        queries[i] = deep_one;
    }
#endif
    int size = queries.size();
    std::cout << "data loaded, size " << queries.size() << std::endl;
    double tavg = 0;
    int max_query_cp_size = 1e7;
    int start_id = 0;

    result_list.resize(size);
    tois.resize(size);

    while (1)
    {
        std::vector<bool> tmp_results;
        std::vector<std::array<std::array<Scalar, 3>, 8>> tmp_queries;
        std::vector<Scalar> tmp_tois;

        int remain = size - start_id;
        double tmp_tall;

        if (remain <= 0)
            break;

        int tmp_nbr = min(remain, max_query_cp_size);
        tmp_results.resize(tmp_nbr);
        tmp_queries.resize(tmp_nbr);
        tmp_tois.resize(tmp_nbr);
        for (int i = 0; i < tmp_nbr; i++)
        {
            tmp_queries[i] = queries[start_id + i];
        }
        all_ccd_run(tmp_queries, is_edge_edge, tmp_results, tmp_tall, tmp_tois, parallel);

        tavg += tmp_tall;
        for (int i = 0; i < tmp_nbr; i++)
        {
            result_list[start_id + i] = tmp_results[i];
            tois[start_id + i] = tmp_tois[i];
        }

        start_id += tmp_nbr;
    }
    tavg /= size;
    std::cout << "avg time " << tavg << std::endl;

    if (expect_list.size() != size)
    {
        std::cout << "size wrong!!!" << std::endl;
        exit(0);
    }
    for (int i = 0; i < size; i++)
    {
        if (expect_list[i])
        {
            total_positives++;
        }
        if (result_list[i] != expect_list[i])
        {
            if (expect_list[i])
            {
                num_false_negatives++;
            }
            else
            {
                num_false_positives++;
            }
        }
    }
    std::cout << "total positives " << total_positives << std::endl;
    std::cout << "num_false_positives " << num_false_positives << std::endl;
    std::cout << "num_false_negatives " << num_false_negatives << std::endl;
    total_number = size;
    if (WRITE_STATISTIC)
    {
        write_summary(
            folder + "method" + std::to_string(int(2021)) + "_is_edge_edge_" + std::to_string(is_edge_edge) + "_" + std::to_string(total_number) + tail + ".csv",
            2021, total_number, total_positives, is_edge_edge,
            num_false_positives, num_false_negatives,
            tavg, time_lower, time_upper);
    }

    if (1)
    {
        std::vector<std::string> titles;
        write_csv(folder + "method" + std::to_string(int(2021)) + "_is_edge_edge_" + std::to_string(is_edge_edge) + "_" +
                      std::to_string(total_number) + "_tois" + tail + ".csv",
                  titles, tois, true);

        // write_csv(folder + "method" + std::to_string(int(2021)) + "_is_edge_edge_" + std::to_string(is_edge_edge) + "_" +
        // std::to_string(total_number) + "_runtime" + tail + ".csv", titles, time_list, true);
    }
}

void run_one_method_over_all_data(const Args &args, int parallel,
                                  const std::string folder = "", const std::string tail = "")
{
    if (args.run_handcrafted_dataset)
    {
        std::cout << "Running handcrafted dataset:\n";
        if (args.run_vf_dataset)
        {
            std::cout << "Vertex-Face:" << std::endl;
            run_rational_data_single_method_parallel(
                args, /*is_edge_edge=*/false, /*is_simu_data=*/false, parallel, folder, tail);
        }
        if (args.run_ee_dataset)
        {
            std::cout << "Edge-Edge:" << std::endl;
            run_rational_data_single_method_parallel(
                args, /*is_edge_edge=*/true, /*is_simu_data=*/false, parallel, folder, tail);
        }
    }
    if (args.run_simulation_dataset)
    {
        std::cout << "Running simulation dataset:\n";
        if (args.run_vf_dataset)
        {
            std::cout << "Vertex-Face:" << std::endl;
            run_rational_data_single_method_parallel(
                args, /*is_edge_edge=*/false, /*is_simu_data=*/true, parallel, folder, tail);
        }
        if (args.run_ee_dataset)
        {
            std::cout << "Edge-Edge:" << std::endl;
            run_rational_data_single_method_parallel(
                args, /*is_edge_edge=*/true, /*is_simu_data=*/true, parallel, folder, tail);
            
        }
    }
}
void run_ours_float_for_all_data(int parallel)
{
    std::string folder = std::string(getenv("HOME")) + "/data0809/"; // this is the output folder
    std::string tail = "_prl_" + std::to_string(parallel);
    Args arg;
    arg.data_dir = std::string(getenv("HOME")) + "/float_with_gt/";

    arg.minimum_separation = 0;
    arg.tight_inclusion_tolerance = 1e-6;
    arg.tight_inclusion_max_iter = 1e6;

    arg.run_ee_dataset = true;
    arg.run_vf_dataset = false;
    
    arg.run_simulation_dataset = true;
    arg.run_handcrafted_dataset = false;

    run_one_method_over_all_data(arg, parallel, folder, tail);

}
int main(int argc, char **argv)
{
    // int deviceCount;
    //     hipGetDeviceCount(&deviceCount);
    //     for(int i=0;i<deviceCount;i++)
    //     {
    //         hipDeviceProp_t devProp;
    //         hipGetDeviceProperties(&devProp, i);
    //         std::cout << "使用GPU device " << i << ": " << devProp.name << std::endl;
    //         std::cout << "设备全局内存总量： " << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
    //         std::cout << "SM的数量：" << devProp.multiProcessorCount << std::endl;
    //         std::cout << "每个线程块的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
    //         std::cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock << std::endl;
    //         std::cout << "设备上一个线程块（Block）种可用的32位寄存器数量： " << devProp.regsPerBlock << std::endl;
    //         std::cout << "每个EM的最大线程数：" << devProp.maxThreadsPerMultiProcessor << std::endl;
    //         std::cout << "每个EM的最大线程束数：" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
    //         std::cout << "设备上多处理器的数量： " << devProp.multiProcessorCount << std::endl;
    //         std::cout << "======================================================" << std::endl;

    //     }
    //     return 0;
    // int alpha=5;
    // double* a = new double[alpha];
    int parallel = 0;
    if (argc == 1)
    {
        parallel = 1;
    }
    else
    {
        parallel = std::stoi(argv[1]);
    }
    if (parallel <= 0)
    {
        std::cout << "wrong parallel nbr = " << parallel << std::endl;
        return 0;
    }

    run_ours_float_for_all_data(parallel);
    std::cout << "done!" << std::endl;
    return 0;
}
