#include "hip/hip_runtime.h"
#include<gputi/queue.h>
#include<iostream>

using namespace std;
  

__device__ item::item(int i) {
	key = i;
}
__device__ item::item() {

}

__device__ item item_max() {
	item it;
	it.key = INT_MAX;
	return it;
}
__device__ item item_min() {
	item it;
	it.key = INT_MIN;
	return it;
}
__device__ bool custom_compare_no_larger(const item &a, const item &b) {
	if (a.key <= b.key) {
		return true;
	}
	return false;
}
__device__ bool custom_compare_less(const item &a, const item &b) {
	if (a.key < b.key) {
		return true;
	}
	return false;
}

// Prototype of a utility function to swap two integers
__device__ void swap(item *x, item *y);



// Constructor: Builds a heap from a given array a[] of given size
// MinHeap::MinHeap(int cap)
// {
// 	heap_size = 0;
// 	capacity = cap;
// 	harr = new item[cap];
// }
__device__ MinHeap::MinHeap()
{
	heap_size = 0;
	capacity = HEAP_SIZE;
	//harr = new item[cap];
}

// Inserts a new key 'k'
__device__ void MinHeap::insertKey(item k)
{	// to avoid overflow, instead of comparing with capacity, we compare with capacity -1
	if (heap_size == capacity-1)
	{
		//cout << "\nOverflow: Could not insertKey\n";
		return;
	}

	// First insert the new key at the end
	heap_size++;
	int i = heap_size - 1;
	harr[i] = k;

	// Fix the min heap property if it is violated
	while (i != 0 && !custom_compare_no_larger(harr[parent(i)], harr[i]))
	{
		swap(&harr[i], &harr[parent(i)]);
		i = parent(i);
	}
}

// Decreases value of key at index 'i' to new_val. It is assumed that
// new_val is smaller than harr[i].
__device__ void MinHeap::decreaseKey(int i, item new_val)
{
	harr[i] = new_val;
	while (i != 0 && !custom_compare_no_larger(harr[parent(i)], harr[i]))
	{
		swap(&harr[i], &harr[parent(i)]);
		i = parent(i);
	}
}

// Method to remove minimum element (or root) from min heap
__device__ item MinHeap::extractMin()
{
	if (heap_size <= 0)
		return item_max();
	if (heap_size == 1)
	{
		heap_size--;
		return harr[0];
	}

	// Store the minimum value, and remove it from heap
	item root = harr[0];
	harr[0] = harr[heap_size - 1];
	heap_size--;
	MinHeapify(0);

	return root;
}


// This function deletes key at index i. It first reduced value to minus
// infinite, then calls extractMin()
__device__ void MinHeap::deleteKey(int i)
{
	decreaseKey(i, item_min());
	extractMin();
}

// A recursive method to heapify a subtree with the root at given index
// This method assumes that the subtrees are already heapified
__device__ void MinHeap::MinHeapify(int i)
{
	int l = left(i);
	int r = right(i);
	int smallest = i;
	if (l < heap_size && custom_compare_less(harr[l], harr[i]))
		smallest = l;
	if (r < heap_size && custom_compare_less(harr[r], harr[smallest]))
		smallest = r;
	if (smallest != i)
	{
		swap(&harr[i], &harr[smallest]);
		MinHeapify(smallest);
	}
}

// A utility function to swap two elements
__device__ void swap(item *x, item *y)
{
	item temp = *x;
	*x = *y;
	*y = temp;
}