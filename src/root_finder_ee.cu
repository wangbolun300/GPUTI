#include "hip/hip_runtime.h"
#include <gputi/root_finder.h>
#include <gputi/queue.h>
namespace ccd{
__device__ void compute_edge_edge_tolerance(const CCDdata &data_in,const CCDConfig& config, CCDOut& out){
    Scalar p000[3], p001[3], p011[3], p010[3], p100[3], p101[3], p111[3], p110[3];
    for(int i=0;i<3;i++){
        p000[i] = data_in.v0s[i] - data_in.v2s[i]; 
        p001[i] = data_in.v0s[i] - data_in.v3s[i];
        p011[i] = data_in.v1s[i] - data_in.v3s[i]; 
        p010[i] = data_in.v1s[i] - data_in.v2s[i];
        p100[i] = data_in.v0e[i] - data_in.v2e[i];
        p101[i] = data_in.v0e[i] - data_in.v3e[i];
        p111[i] = data_in.v1e[i] - data_in.v3e[i];
        p110[i] = data_in.v1e[i] - data_in.v2e[i];
    }
    Scalar dl=0;
    for(int i=0;i<3;i++){
        dl=max(dl,fabs(p100[i]-p000[i]));
        dl=max(dl,fabs(p101[i]-p001[i])); 
        dl=max(dl,fabs(p111[i]-p011[i]));
        dl=max(dl,fabs(p110[i]-p010[i]));
    }
    dl*=3;
    out.tol[0] = config.co_domain_tolerance / dl;

    dl=0;
    for(int i=0;i<3;i++){
        dl=max(dl,fabs(p010[i]-p000[i]));
        dl=max(dl,fabs(p110[i]-p100[i])); 
        dl=max(dl,fabs(p111[i]-p101[i]));
        dl=max(dl,fabs(p011[i]-p001[i]));
    }
    dl*=3;
    out.tol[1] = config.co_domain_tolerance / dl;
    
    dl=0;
    for(int i=0;i<3;i++){
        dl=max(dl,fabs(p001[i]-p000[i]));
        dl=max(dl,fabs(p101[i]-p100[i])); 
        dl=max(dl,fabs(p111[i]-p110[i]));
        dl=max(dl,fabs(p011[i]-p010[i]));
    }
    dl*=3;
    out.tol[2] = config.co_domain_tolerance / dl;
}

__device__ __host__ void get_numerical_error_ee(
    const CCDdata &data_in,
    BoxCompute &box)
{
    Scalar vffilter;

#ifdef GPUTI_USE_DOUBLE_PRECISION
    vffilter = 6.217248937900877e-15;
#else
    vffilter = 3.337861e-06;
#endif
    Scalar xmax = fabs(data_in.v0s[0]);
    Scalar ymax = fabs(data_in.v0s[1]);
    Scalar zmax = fabs(data_in.v0s[2]);

    xmax = max(xmax,fabs(data_in.v1s[0]));
    ymax = max(ymax,fabs(data_in.v1s[1]));
    zmax = max(zmax,fabs(data_in.v1s[2]));
    
    xmax = max(xmax,fabs(data_in.v2s[0]));
    ymax = max(ymax,fabs(data_in.v2s[1]));
    zmax = max(zmax,fabs(data_in.v2s[2]));

    xmax = max(xmax,fabs(data_in.v3s[0]));
    ymax = max(ymax,fabs(data_in.v3s[1]));
    zmax = max(zmax,fabs(data_in.v3s[2]));

    xmax = max(xmax,fabs(data_in.v0e[0]));
    ymax = max(ymax,fabs(data_in.v0e[1]));
    zmax = max(zmax,fabs(data_in.v0e[2]));

    xmax = max(xmax,fabs(data_in.v1e[0]));
    ymax = max(ymax,fabs(data_in.v1e[1]));
    zmax = max(zmax,fabs(data_in.v1e[2]));

    xmax = max(xmax,fabs(data_in.v2e[0]));
    ymax = max(ymax,fabs(data_in.v2e[1]));
    zmax = max(zmax,fabs(data_in.v2e[2]));

    xmax = max(xmax,fabs(data_in.v3e[0]));
    ymax = max(ymax,fabs(data_in.v3e[1]));
    zmax = max(zmax,fabs(data_in.v3e[2]));

    xmax = max(xmax, Scalar(1));
    ymax = max(ymax, Scalar(1));
    zmax = max(zmax, Scalar(1));

    box.err[0] = xmax * xmax * xmax * vffilter;
    box.err[1] = ymax * ymax * ymax * vffilter;
    box.err[2] = zmax * zmax * zmax * vffilter;
    return;
}
__device__ Scalar calculate_ee(const CCDdata &data_in, const BoxPrimatives& bp){
    Scalar edge0_vertex0 = (data_in.v0e[bp.dim] - data_in.v0s[bp.dim]) * bp.t + data_in.v0s[bp.dim];
    Scalar edge0_vertex1 = (data_in.v1e[bp.dim] - data_in.v1s[bp.dim]) * bp.t + data_in.v1s[bp.dim];
    Scalar edge1_vertex0 = (data_in.v2e[bp.dim] - data_in.v2s[bp.dim]) * bp.t + data_in.v2s[bp.dim];
    Scalar edge1_vertex1 = (data_in.v3e[bp.dim] - data_in.v3s[bp.dim]) * bp.t + data_in.v3s[bp.dim];
    Scalar result=((edge0_vertex1 - edge0_vertex0) * bp.u+ edge0_vertex0)
                -( (edge1_vertex1 - edge1_vertex0) * bp.v+ edge1_vertex0);
            
    return result;
}

__device__ bool Origin_in_ee_inclusion_function(const CCDdata &data_in, BoxCompute& box, CCDOut& out){
    BoxPrimatives bp;
    Scalar vmin=SCALAR_LIMIT;
    Scalar vmax=-SCALAR_LIMIT;
    Scalar value;
    for(bp.dim=0;bp.dim<3;bp.dim++){
        vmin=SCALAR_LIMIT;
        vmax=-SCALAR_LIMIT;
        for (int i = 0; i < 2; i++)
        {
            for (int j = 0; j < 2; j++)
            {
                for (int k = 0; k < 2; k++)
                {
                    bp.b[0] = i;
                    bp.b[1] = j;
                    bp.b[2] = k; //100
                    bp.calculate_tuv(box);
                    value = calculate_ee(data_in, bp);
                    vmin = min(vmin, value);
                    vmax = max(vmax, value);
                    
                }
            }
        }

        // get the min and max in one dimension
        box.true_tol = max(box.true_tol, vmax - vmin); // this is the real tolerance

        if (vmin > box.err[bp.dim] || vmax < -box.err[bp.dim])
        {
            return false;
        }

        if (vmin < -box.err[bp.dim] || vmax > box.err[bp.dim])
        {
            box.box_in = false;
        }
        
    }
    return true;
}

__device__ void bisect_ee_and_push(BoxCompute& box,const CCDConfig& config, MinHeap& istack,CCDOut& out){
    interval_pair halves(box.current_item.itv[box.split]);// bisected
    bool inserted;
    if (halves.first.first  >= halves.first.second)
    {
        out.overflow_flag = BISECTION_OVERFLOW;
        return;
    }
    if (halves.second.first>= halves.second.second)
    {
        out.overflow_flag = BISECTION_OVERFLOW;
        return;
    }

    if (config.max_t != 1 && box.split == 0)
    {
        if (halves.second.first <= config.max_t)
        {
            box.current_item.itv[box.split] = halves.second;
            inserted = istack.insertKey(item(box.current_item.itv, box.current_item.level + 1));
            if (inserted == false)
            {
                out.overflow_flag = HEAP_OVERFLOW;
            }
        }

        box.current_item.itv[box.split] = halves.first;
        inserted = istack.insertKey(item(box.current_item.itv, box.current_item.level + 1));
        if (inserted == false)
        {
            out.overflow_flag = HEAP_OVERFLOW;
        }
    }
    else
    {
        box.current_item.itv[box.split] = halves.second;
        inserted = istack.insertKey(item(box.current_item.itv, box.current_item.level + 1));
        if (inserted == false)
        {
            out.overflow_flag = HEAP_OVERFLOW;
        }
        box.current_item.itv[box.split] = halves.first;
        inserted = istack.insertKey(item(box.current_item.itv, box.current_item.level + 1));
        if (inserted == false)
        {
            out.overflow_flag = HEAP_OVERFLOW;
        }
    }

}

__device__ void edgeEdgeCCD(const CCDdata &data_in,const CCDConfig& config, CCDOut& out){
    
    MinHeap istack;// now when initialized, size is 1 and initialized with [0,1]^3
    compute_edge_edge_tolerance(data_in, config, out);
    BoxCompute box;

#ifdef CALCULATE_ERROR_BOUND
    get_numerical_error_ee(data_in, box);
#else
    box.err[0] = config.err_in[0];
    box.err[1] = config.err_in[1];
    box.err[2] = config.err_in[2];
#endif

    out.output_tolerance = config.co_domain_tolerance;

    // this is used to catch the tolerance for each level
    Scalar temp_output_tolerance = config.co_domain_tolerance;
    // LINENBR 2
    int refine = 0;
    // temp_toi is to catch the first toi of each level
    Scalar temp_toi = SCALAR_LIMIT;
    Scalar skip_toi =SCALAR_LIMIT;
    
    bool use_skip = false; // when tolerance is small enough or when box in epsilon, this is activated.
    int current_level = -2; // in the begining, current_level != level
    int box_in_level = -2;  // this checks if all the boxes before this
    // level < tolerance. only true, we can return when we find one overlaps eps box and smaller than tolerance or eps-box
    bool this_level_less_tol = true;
    bool find_level_root = false;

    while (!istack.empty())
    {
        if (out.overflow_flag != NO_OVERFLOW)
        {
            break;
        }

        //LINENBR 6
        istack.extractMin(box.current_item); // get the level and the intervals

        // if this box is later than TOI_SKIP in time, we can skip this one.
        // TOI_SKIP is only updated when the box is small enough or totally contained in eps-box
        if (box.current_item.itv[0].first>=skip_toi)
        {
            continue;
        }
        if (box_in_level != box.current_item.level)
        { // before check a new level, set this_level_less_tol=true
            box_in_level = box.current_item.level;
            this_level_less_tol = true;
        }
        // LINENBR 8
        refine++;
        bool zero_in =
            Origin_in_ee_inclusion_function(data_in,box, out);
        
        if (!zero_in)
            continue;

        // get the width of the box
        box.widths[0] = box.current_item.itv[0].second - box.current_item.itv[0].first;
        box.widths[1] = box.current_item.itv[1].second - box.current_item.itv[1].first;
        box.widths[2] = box.current_item.itv[2].second - box.current_item.itv[2].first;
                
        // LINENBR 15, 16
        // Condition 1, stopping condition on t, u and v is satisfied. this is useless now since we have condition 2
        bool condition = box.widths[0] <= out.tol[0] && box.widths[1] <= out.tol[1] && box.widths[2] <= out.tol[2];
        if(condition){
            out.toi=box.current_item.itv[0].first;
            out.result=true;
            return;
        }
        // Condition 2, zero_in = true, box inside eps-box and in this level,
        // no box whose zero_in is true but box size larger than tolerance, can return
        condition = box.box_in && this_level_less_tol;
        if(condition){
            out.toi=box.current_item.itv[0].first;
            out.result= true;
            return;
        }

        bool tol_condition = box.true_tol <= config.co_domain_tolerance;
        if (!tol_condition)
        {
            this_level_less_tol = false;
            // this level has at least one box whose size > tolerance, thus we
            // cannot directly return if find one box whose size < tolerance or box-in
        }

        // Condition 3, in this level, we find a box that zero-in and size < tolerance.
        // and no other boxes whose zero-in is true in this level before this one is larger than tolerance, can return
        condition = this_level_less_tol;
        if(condition){
            out.toi=box.current_item.itv[0].first;
            out.result=true;
            return;
        }

        // This is for early termination, finding the earlist root of this level in case of early termination happens
        if (current_level != box.current_item.level)
        {
            // LINENBR 22
            current_level = box.current_item.level;
            find_level_root = false;
        }
        if (!find_level_root)
        {
            // LINENBR 11
            // this is the first toi of this level
            temp_toi = box.current_item.itv[0].first;
            // if the real tolerance is larger than input, use the real one;
            // if the real tolerance is smaller than input, use input
            temp_output_tolerance = max(box.true_tol,config.co_domain_tolerance);
            find_level_root =true; // this ensures always find the earlist root
        }

        // LINENBR 12
        if (refine > config.max_itr)
        {
            out.overflow_flag = ITERATION_OVERFLOW;
            break;
        }

        // if this box is small enough, or inside of eps-box, then just continue,
        // but we need to record the collision time
        if (tol_condition || box.box_in )
        {
            if(box.current_item.itv[0].first<skip_toi)
            {
                skip_toi=box.current_item.itv[0].first;
            }
            use_skip = true;
            continue;
        }
        split_dimension(out,box);
        bisect_ee_and_push(box,config, istack,out);
    }
    if (out.overflow_flag != NO_OVERFLOW)
    {
        out.toi = temp_toi;
        out.output_tolerance = temp_output_tolerance;
        out.result=true;
        return;
    }

    if (use_skip)
    {
        out.toi = skip_toi;
        out.result=true;
        return;
    }
    out.result=false;
    return;
}



}// namespace ccd